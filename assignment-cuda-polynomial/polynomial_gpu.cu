#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#define M 512

__device__ float polynomial (float x, float* poly, int degree) {
  float out = 0.;
  float xtothepowerof = 1.;
  for (int i=0; i<=degree; ++i) {
    out += xtothepowerof*poly[i];
    xtothepowerof *= x;
  }
  return out;
}

__global__ void polynomial_expansion (float* poly, int degree,
			   int n, float* array) {

  int index = threadIdx.x + blockIdx.x*blockDim.x;
  
    if(index<n)
    {
      array[i] = polynomial (array[i], poly, degree);  
    }

}


int main (int argc, char* argv[]) {
  //TODO: add usage
  
  if (argc < 3) {
     std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
     return -1;
  }

  int n = atoi(argv[1]); //TODO: atoi is an unsafe function
  int degree = atoi(argv[2]);
  int nbiter = 1;

  float* array,*poly;
  int size_array = n*sizeof(float);
  int size_poly = (degree+1)*sizeof(float);

  float* d_array,*d_poly;

  array = (float *)malloc(size_array);
  poly = (float *)malloc(size_poly);

  hipMalloc((void **)&d_array,size_array);
  hipMalloc((void **)&d_poly,size_poly;


  for (int i=0; i<n; ++i)
    array[i] = 1.;

  for (int i=0; i<degree+1; ++i)
    poly[i] = 1.;

  hipMemcpy(d_array,array,size_array,hipMemcpyHostToDevice);
  hipMemcpy(d_poly,poly,size_poly,hipMemcpyHostToDevice);
  
  std::chrono::time_point<std::chrono::system_clock> begin, end;
  begin = std::chrono::system_clock::now();
  
  for (int iter = 0; iter<nbiter; ++iter)
    polynomial_expansion<<<(n+M-1)/M,M>>>(poly, degree, n, array);

  hipMemcpy(array,d_array,size_array,hipMemcpyDeviceToHost);

  end = std::chrono::system_clock::now();
  std::chrono::duration<double> totaltime = (end-begin)/nbiter;

  std::cerr<<array[0]<<std::endl;
  std::cout<<n<<" "<<degree<<" "<<totaltime.count()<<std::endl;

  free(array);
  free(poly);

  hipFree(d_array);
  hipFree(d_poly);

  return 0;
}
